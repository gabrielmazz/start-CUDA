#include <cstdio>
#include <vector>
#include <hip/hip_runtime.h>

// Kernel na GPU: soma elemento a elemento
__global__ void somaVetoresKernel(const int* a, const int* b, int* c, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) c[i] = a[i] + b[i];
}

int main() {
    // ---- Dados no host (CPU) ----
    int n = 64;                         // Tamanho do vetor
    std::vector<int> a(n), b(n), c(n);  // Vetores de entrada e saída

    // Inicialização dos vetores
    for (int i = 1; i < n; ++i) {
        a[i] = i;
        b[i] = 1000 + i;
    }

    // ---- Alocar no device (GPU) ----
    int *d_a = nullptr, *d_b = nullptr, *d_c = nullptr;
    size_t bytes = n * sizeof(int);
    hipMalloc(&d_a, bytes);
    hipMalloc(&d_b, bytes);
    hipMalloc(&d_c, bytes);

    // ---- Copiar CPU -> GPU ----
    hipMemcpy(d_a, a.data(), bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b.data(), bytes, hipMemcpyHostToDevice);

    // ---- Configurar e lançar kernel ----
    int threads = 256;                       // threads por bloco
    int blocks  = (n + threads - 1) / threads; // nº de blocos
    somaVetoresKernel<<<blocks, threads>>>(d_a, d_b, d_c, n);
    hipDeviceSynchronize();

    // ---- Copiar GPU -> CPU ----
    hipMemcpy(c.data(), d_c, bytes, hipMemcpyDeviceToHost);

    // ---- Conferir resultado ----
    for (int i = 0; i < n; ++i) {
        printf("c[%d] = %d (%d + %d)\n", i, c[i], a[i], b[i]);
    }

    // ---- Liberar ----
    hipFree(d_a); hipFree(d_b); hipFree(d_c);
    return 0;
}
